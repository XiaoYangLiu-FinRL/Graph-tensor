/*#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "based.h"
#include "gFT.h"
#include "gIFT.h"
*/
#include "gProduct.h"

/*
 * INPUT:
 * @param: G1 m*r*k
 * @param: G2 r*n*k
 * OUTPUT:
 * @param: G3 m*n*k
*/

void gProduct_3D_batched(float* d_U, int k, float* d_G1, int m, int r, float* d_G2, int n, float* d_G3){
	//printTensor_d(d_G1, m, r, k, "G1");
	//printTensor_d(d_G2, r, n, k, "G2");
	float* d_G1_f = NULL;
	hipMalloc((void**)&d_G1_f,sizeof(float)*m*r*k);
	gFT_3D_batched_d(d_U, k, d_G1, m, r, d_G1_f);
	
	float* d_G2_f = NULL;
	hipMalloc((void**)&d_G2_f,sizeof(float)*r*n*k);
	gFT_3D_batched_d(d_U, k, d_G2, r, n, d_G2_f);

	float* d_G3_f = NULL;
	hipMalloc((void**)&d_G3_f,sizeof(float)*m*n*k);
	tensorMultiplytensor_d(d_G1_f, 0, m, r, d_G2_f, n, k, d_G3_f);

	gIFT_3D_batched_d(d_U, k, d_G3_f, m, n, d_G3);

    hipFree(d_G1_f);
    hipFree(d_G2_f);
    hipFree(d_G3_f);
}

void gProduct_3D_based(float* d_U, int k, float* d_G1, int m, int r, float* d_G2, int n, float* d_G3){
	//printTensor_d(d_G1, m, r, k, "G1");
	//printTensor_d(d_G2, r, n, k, "G2");
	float* d_G1_f = NULL;
	hipMalloc((void**)&d_G1_f,sizeof(float)*m*r*k);
	gFT_3D_based_d(d_U, k, d_G1, m, r, d_G1_f);
	float* d_G2_f = NULL;
	hipMalloc((void**)&d_G2_f,sizeof(float)*r*n*k);
	gFT_3D_based_d(d_U, k, d_G2, r, n, d_G2_f);

	float* d_G3_f = NULL;
	hipMalloc((void**)&d_G3_f,sizeof(float)*m*n*k);
	//tensorMultiplytensor_d(d_G1_f, 0, m, r, d_G2_f, n, k, d_G3_f);
	hipblasHandle_t handle;
	int Am = m;
	int An = r;
	int Bn = n;
	int Bm = r;
	int strA = Am*An;
	int strB = Bm*Bn;
	int strC = Am*Bn;
	hipblasCreate(&handle);
	float alpha = 1;
	float beta = 0;
	for(int i=0; i<k; i++){
	if(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Am, Bn, Bm,
	        &alpha, d_G1_f+i*strA, Am,d_G2_f+i*strB, Bm,  &beta,
	        d_G3_f+i*strC, Am) !=HIPBLAS_STATUS_SUCCESS){
	
		fprintf(stdout,"[%s]:[%d] CUFFT ERROR: hipblasCgemm failed!",__FUNCTION__,__LINE__);
		return;
	}
	}
	hipblasDestroy(handle);

	gIFT_3D_based_d(d_U, k, d_G3_f, m, n, d_G3);

    hipFree(d_G1_f);
    hipFree(d_G2_f);
    hipFree(d_G3_f);
}

void gProduct_3D(float* d_U, int k, float* d_G1, int m, int r, float* d_G2, int n, float* d_G3){
	float* d_G2_f = NULL;
	hipMalloc((void**)&d_G2_f,sizeof(float)*r*n*k);
	gFT_3D_batched_d(d_U, k, d_G2, r, n, d_G2_f);
	tensorMultiplytensor_d(d_G1, 0, m, r, d_G2_f, n, k, d_G3);
	hipFree(d_G2_f);
}
